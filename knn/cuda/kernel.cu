
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void calcDistances(double *data, double *target, double *distances, long long n, int dim)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    {
        for (int i = idx; i < n; i += blockDim.x * gridDim.x)
        {
            double sum = 0;
            for (int j = 0; j < dim; j++)
            {
                sum += (data[i * dim + j] - target[j]) * (data[i * dim + j] - target[j]);
            }
            distances[i] = sqrt(sum);
        }
    }
}

__device__ double euclidean_distance(double *data, int idx, double *target, int dim)
{
    double sum = 0;
    for (int i = 0; i < dim; i++)
    {
        sum += (data[idx * dim + i] - target[i]) * (data[idx * dim + i] - target[i]);
    }
    return sqrt(sum);
}

__device__ void sortElements(double *data, double *distances, double *distancesOut, long long startElement,
                             long long endElement, long long startElementCopy, long long endElementCopy, double *target,
                             int k, int *nearesrtNeighborsIdxs, int dim)
{
    // Initialize nearestNeighborsIdxs and distancesOut with initial k values
    for (int i = 0; i < k; i++)
    {
        nearesrtNeighborsIdxs[i] = startElement + i;
        distancesOut[i + startElementCopy] = distances[startElement + i];
    }

    // Sort distances and update nearestNeighborsIdxs
    for (long long i = startElement + k; i <= endElement; i++)
    {
        // Find the index of the maximum distance in distancesOut
        long long maxIdx = 0;
        double maxVal = distancesOut[startElementCopy];
        for (long long j = 1; j < k; j++)
        {
            if (distancesOut[j + startElementCopy] > maxVal)
            {
                maxIdx = j;
                maxVal = distancesOut[j + startElementCopy];
            }
        }

        // Update distancesOut and nearestNeighborsIdxs if the current distance is smaller
        if (distances[i] < maxVal)
        {
            distancesOut[maxIdx + startElementCopy] = distances[i];
            nearesrtNeighborsIdxs[maxIdx] = i;
        }
    }
}

__global__ void knn(double *data, int *labels, double *distances, int threadSize, long long n, int dim, int k,
                    double *target, double *output, int *labelsOutput, double *distancesOut)
{
    // print thread info
    long long startElement = (blockIdx.x * blockDim.x + threadIdx.x) * threadSize;
    long long endElement = startElement + threadSize < n ? startElement + threadSize - 1 : n - 1;
    long long totalElements = endElement - startElement + 1;
    k = k < totalElements ? k : totalElements;

    if (startElement < n)
    {
        // copy elements to data again
        long long startElementCopy = (blockIdx.x * blockDim.x + threadIdx.x) * k;
        long long endElementCopy = startElementCopy + k - 1;

        int *nearesrtNeighborsIdxs = (int *)malloc(sizeof(int) * k);
        sortElements(data, distances, distancesOut, startElement, endElement, startElementCopy, endElementCopy, target,
                     k, nearesrtNeighborsIdxs, dim);

        for (long long i = startElementCopy; i <= endElementCopy; i++)
        {
            for (int j = 0; j < dim; j++)
            {
                output[i * dim + j] = data[nearesrtNeighborsIdxs[i - startElementCopy] * dim + j];
            }
            labelsOutput[i] = labels[nearesrtNeighborsIdxs[i - startElementCopy]];
        }
        free(nearesrtNeighborsIdxs);
    }
}