#include "hip/hip_runtime.h"
#include "./knn.h"

int k = 0, n = 0, dim = 0;

void bubbleSortResult(double *output, double *target)
{
    for (int i = 0; i < k; i++)
    {
        for (int j = i + 1; j < k; j++)
        {
            double dist1 = 0;
            double dist2 = 0;
            for (int l = 0; l < dim; l++)
            {
                dist1 += (output[i * dim + l] - target[l]) * (output[i * dim + l] - target[l]);
                dist2 += (output[j * dim + l] - target[l]) * (output[j * dim + l] - target[l]);
            }
            if (dist1 > dist2)
            {
                for (int l = 0; l < dim; l++)
                {
                    double temp = output[i * dim + l];
                    output[i * dim + l] = output[j * dim + l];
                    output[j * dim + l] = temp;
                }
            }
        }
    }

    // print sorted output
    cout << "Sorted output:" << endl;
    for (int i = 0; i < k; i++)
    {
        cout << "Data " << i << ": ";
        for (int j = 0; j < dim; j++)
        {
            cout << fixed;
            cout.precision(10);
            cout << output[i * dim + j] << " ";
        }

        cout << "Distance: ";
        double dist = 0;
        for (int j = 0; j < dim; j++)
        {
            dist += (output[i * dim + j] - target[j]) * (output[i * dim + j] - target[j]);
        }
        cout << fixed;
        cout.precision(10);
        cout << sqrt(dist);
        cout << endl;
    }
}

int main(int argc, char **argv)
{
    if (argc != 3)
    {
        cout << "Usage: ./knn input_file output_file" << endl;
        return 1;
    }
    string input_file = argv[1];
    string output_file = argv[2];

    double *data = NULL;
    int *labels = NULL;
    double *target = NULL;

    // Read data
    read_data(input_file, data, labels, target);

    int threadSize = 2 * k;

    // Print top 5 data
    // print_top(data, labels, n, target);

    // allocate device memory
    double *d_data, *d_data2, *d_target, *d_distances, *d_distances2;
    int *d_labels, *d_labels2;

    hipMalloc(&d_data, sizeof(double) * n * dim);
    hipMalloc(&d_labels, sizeof(int) * n);
    hipMalloc(&d_target, sizeof(double) * dim);
    hipMalloc(&d_data2, sizeof(double) * n * dim);
    hipMalloc(&d_labels2, sizeof(int) * n);
    hipMalloc(&d_distances, sizeof(double) * n);
    hipMalloc(&d_distances2, sizeof(double) * n);

    // copy data to device
    hipMemcpy(d_data, data, sizeof(double) * n * dim, hipMemcpyHostToDevice);
    hipMemcpy(d_labels, labels, sizeof(int) * n, hipMemcpyHostToDevice);
    hipMemcpy(d_target, target, sizeof(double) * dim, hipMemcpyHostToDevice);

    // calculate distances
    long long calcDistThreadSize = 256;
    calcDistances<<<(n + calcDistThreadSize - 1) / calcDistThreadSize, calcDistThreadSize>>>(d_data, d_target,
                                                                                             d_distances, n, dim);

    // call kernel
    long long i = 0;
    while (n > k)
    {
        int threadsPerBlock = 256;
        long long blocksPerGrid = (n + threadSize - 1) / threadSize;

        if (i % 2 == 0)
        {
            knn<<<blocksPerGrid, threadsPerBlock>>>(d_data, d_labels, d_distances, threadSize, n, dim, k, d_target,
                                                    d_data2, d_labels2, d_distances2);
        }
        else
        {
            knn<<<blocksPerGrid, threadsPerBlock>>>(d_data2, d_labels2, d_distances2, threadSize, n, dim, k, d_target,
                                                    d_data, d_labels, d_distances);
        }
        i++;
        long long numKs = n / (threadSize);
        long long rem = n % (threadSize);
        n = numKs * k;
        if (rem < k)
        {
            n += rem;
        }
        else
        {
            n += k;
        }
        cout << "n: " << n << endl;
        // wait for kernel to finish
        hipDeviceSynchronize();
    }

    // check for errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess)
    {
        cout << "Error: " << hipGetErrorString(error) << endl;
        return 1;
    }

    // allocate memory for output
    double *output = (double *)malloc(sizeof(double) * k * dim);
    int *labelsOutput = (int *)malloc(sizeof(int) * k);

    // copy output from device to host
    if (i % 2 == 0)
    {
        hipMemcpy(output, d_data, sizeof(double) * k * dim, hipMemcpyDeviceToHost);
        hipMemcpy(labelsOutput, d_labels, sizeof(int) * k, hipMemcpyDeviceToHost);
    }
    else
    {
        hipMemcpy(output, d_data2, sizeof(double) * k * dim, hipMemcpyDeviceToHost);
        hipMemcpy(labelsOutput, d_labels2, sizeof(int) * k, hipMemcpyDeviceToHost);
    }

    // sort output
    bubbleSortResult(output, target);

    // free device memory
    hipFree(d_data);
    hipFree(d_labels);

    // Write output
    write_data(output_file, output, labelsOutput);

    // Free memory
    free(data);
    free(labels);
    free(target);
    free(output);
    free(labelsOutput);

    return 0;
}