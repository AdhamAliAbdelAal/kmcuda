#include "hip/hip_runtime.h"
#include <iostream>
using namespace std;

int k, n, dim;

bool read_data(string input_file, double *&data, int *&labels,
               double *&target) {
  FILE *file = freopen(input_file.c_str(), "r", stdin);
  if (file == NULL) {
    cout << "Cannot open file " << input_file << endl;
    return false;
  }
  cout << "Reading data from " << input_file << endl;
  cin >> k >> n >> dim;
  cout << "k = " << k << ", n = " << n << ", dim = " << dim << endl;
  data = (double *)malloc(sizeof(double) * n * dim);
  labels = (int *)malloc(sizeof(int) * n);
  target = (double *)malloc(sizeof(double) * dim);

  for (int i = 0; i < dim; i++) {
    cin >> target[i];
  }

  for (int i = 0; i < n; i++) {
    for (int j = 0; j < dim; j++) {
      cin >> data[i * dim + j];
    }
    cin >> labels[i];
  }
  fclose(file);
  return true;
}

bool write_data(string output_file, double *output) {
  FILE *file = freopen(output_file.c_str(), "w", stdout);
  if (file == NULL) {
    cout << "Cannot open file " << output_file << endl;
    return false;
  }
  for (int i = 0; i < k; i++) {
    for (int j = 0; j < dim; j++) {
      cout << output[i * dim + j] << " ";
    }
    cout << endl;
  }
  fclose(file);
  return true;
}

void print_top(double *data, int *labels, int n, double *target) {
  cout << "Target data: ";
  for (int i = 0; i < dim; i++) {
    cout << target[i] << " ";
  }
  cout << endl;
  cout << "Top " << n << " data:" << endl;
  for (int i = 0; i < n; i++) {
    cout << "Data " << i << ": ";
    for (int j = 0; j < dim; j++) {
      cout << data[i * dim + j] << " ";
    }
    cout << "Label: " << labels[i] << endl;
  }
}

__device__ double euclidean_distance(double *data, int idx, double *target,
                                     int dim) {
  double sum = 0;
  printf("element %d: (", idx);
  for (int i = 0; i < dim; i++) {
    // print element
    printf("%f, ", data[idx * dim + i]);
    sum +=
        (data[idx * dim + i] - target[i]) * (data[idx * dim + i] - target[i]);
  }
  printf(", sum = %f)\n", sqrt(sum));
  return sqrt(sum);
}

__device__ void sortElements(double *data, int startElement, int endElement,
                             double *target, int k, int *nearesrtNeighborsIdxs,
                             int dim) {
  double *distances = (double *)malloc(sizeof(double) * k);
  for (int i = startElement; i <= endElement; i++) {
    double dist = euclidean_distance(data, i, target, dim);
    if (i < k) {
      distances[i] = dist;
      nearesrtNeighborsIdxs[i] = i;
    } else {
      int maxIdx = 0;
      for (int j = 1; j < k; j++) {
        if (distances[j] > distances[maxIdx]) {
          maxIdx = j;
        }
      }
      if (dist < distances[maxIdx]) {
        distances[maxIdx] = dist;
        nearesrtNeighborsIdxs[maxIdx] = i;
      }
    }
  }
}

__global__ void knn(double *data, int *labels, int threadSize, int n, int dim,
                    int k, double *target) {
  // print thread info
  printf("Thread %d %d\n", threadIdx.x, blockIdx.x);

  int startElement = (blockIdx.x * blockDim.x + threadIdx.x) * threadSize;
  int endElement =
      startElement + threadSize < n ? startElement + threadSize : n - 1;
  // pritn start and end element
  printf("Start element: %d, End element: %d\n", startElement, endElement);
  int *nearesrtNeighborsIdxs = (int *)malloc(sizeof(int) * k);
  sortElements(data, startElement, endElement, target, k, nearesrtNeighborsIdxs,
               dim);
  // DEBUG print nearest neighbors
  for (int i = 0; i < k; i++) {
    printf("Nearest neighbor %d: %d\n", i, nearesrtNeighborsIdxs[i]);
  }

  // copy elements to data again
  int startElementCopy = (blockIdx.x * blockDim.x + threadIdx.x) * k;
  int endElementCopy = startElementCopy + k < n ? startElementCopy + k : n - 1;
  for (int i = startElementCopy; i <= endElementCopy; i++) {
    for (int j = 0; j < dim; j++) {
      data[i * dim + j] = data[nearesrtNeighborsIdxs[i] * dim + j];
    }
  }
  free(nearesrtNeighborsIdxs);
}

int main(int argc, char **argv) {
  if (argc != 3) {
    cout << "Usage: ./knn input_file output_file" << endl;
    return 1;
  }
  string input_file = argv[1];
  string output_file = argv[2];

  double *data = NULL;
  int *labels = NULL;
  double *target = NULL;

  // Read data
  read_data(input_file, data, labels, target);

  // Print top 5 data
  // print_top(data, labels, n, target);

  // allocate device memory
  double *d_data;
  int *d_labels;
  double *d_target;
  hipMalloc(&d_data, sizeof(double) * n * dim);
  hipMalloc(&d_labels, sizeof(int) * n);
  hipMalloc(&d_target, sizeof(double) * dim);

  // copy data to device
  hipMemcpy(d_data, data, sizeof(double) * n * dim, hipMemcpyHostToDevice);
  hipMemcpy(d_labels, labels, sizeof(int) * n, hipMemcpyHostToDevice);
  hipMemcpy(d_target, target, sizeof(double) * dim, hipMemcpyHostToDevice);

  // call kernel
  // int threadSize = 2 * k;
  int threadSize = n;
  knn<<<1, 1>>>(d_data, d_labels, threadSize, n, dim, k, d_target);

  // wait for kernel to finish
  hipDeviceSynchronize();

  // check for errors
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    cout << "Error: " << hipGetErrorString(error) << endl;
    return 1;
  }

  // allocate memory for output
  double *output = (double *)malloc(sizeof(double) * k * dim);

  // copy output from device to host
  hipMemcpy(output, d_data, sizeof(double) * k * dim, hipMemcpyDeviceToHost);

  // free device memory
  hipFree(d_data);
  hipFree(d_labels);

  // Write output
  write_data(output_file, output);

  // Free memory
  free(data);

  return 0;
}