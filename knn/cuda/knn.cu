
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

int k, n, dim;

bool read_data(string input_file, double*& data, int*& labels,
               double*& target) {
  FILE* file = freopen(input_file.c_str(), "r", stdin);
  if (file == NULL) {
    cout << "Cannot open file " << input_file << endl;
    return false;
  }
  cout << "Reading data from " << input_file << endl;
  cin >> k >> n >> dim;
  cout << "k = " << k << ", n = " << n << ", dim = " << dim << endl;
  data = (double*)malloc(sizeof(double) * n * dim);
  labels = (int*)malloc(sizeof(int) * n);
  target = (double*)malloc(sizeof(double) * dim);

  for (int i = 0; i < dim; i++) {
    cin >> target[i];
  }

  for (int i = 0; i < n; i++) {
    for (int j = 0; j < dim; j++) {
      cin >> data[i * dim + j];
    }
    cin >> labels[i];
  }
  fclose(file);
  return true;
}

bool write_data(string output_file, double* output) {
  FILE* file = freopen(output_file.c_str(), "w", stdout);
  if (file == NULL) {
    cout << "Cannot open file " << output_file << endl;
    return false;
  }
  for (int i = 0; i < k; i++) {
    for (int j = 0; j < dim; j++) {
      cout << output[i * dim + j] << " ";
    }
    cout << endl;
  }
  fclose(file);
  return true;
}

void print_top(double* data, int* labels, int n, double* target) {
  cout << "Target data: ";
  for (int i = 0; i < dim; i++) {
    cout << target[i] << " ";
  }
  cout << endl;
  cout << "Top " << n << " data:" << endl;
  for (int i = 0; i < n; i++) {
    cout << "Data " << i << ": ";
    for (int j = 0; j < dim; j++) {
      cout << data[i * dim + j] << " ";
    }
    cout << "Label: " << labels[i] << endl;
  }
}

__global__ void knn(double* data, int* labels, int n, int dim, double* output,
                    int k, double* target) {
  // print thread info
  printf("Thread %d %d\n", threadIdx.x, blockIdx.x);

  // print top 5 data
  for (int i = 0; i < 5; i++) {
    printf("Data %d: ", i);
    for (int j = 0; j < dim; j++) {
      printf("%f ", data[i * dim + j]);
    }
    printf("Label: %d\n", labels[i]);
  }
  printf("Finished printing top 5 data\n");
}

int main(int argc, char** argv) {
  if (argc != 3) {
    cout << "Usage: ./knn input_file output_file" << endl;
    return 1;
  }
  string input_file = argv[1];
  string output_file = argv[2];

  double* data = NULL;
  int* labels = NULL;
  double* target = NULL;

  // Read data
  read_data(input_file, data, labels, target);

  // Print top 5 data
  print_top(data, labels, n, target);

  // allocate device memory
  double* d_data;
  int* d_labels;
  double* d_output;
  double* d_target;
  hipMalloc(&d_data, sizeof(double) * n * dim);
  hipMalloc(&d_labels, sizeof(int) * n);
  hipMalloc(&d_output, sizeof(double) * k * dim);
  hipMalloc(&d_target, sizeof(double) * dim);

  // copy data to device
  hipMemcpy(d_data, data, sizeof(double) * n * dim, hipMemcpyHostToDevice);
  hipMemcpy(d_labels, labels, sizeof(int) * n, hipMemcpyHostToDevice);
  hipMemcpy(d_output, target, sizeof(double) * k * dim,
             hipMemcpyHostToDevice);

  // call kernel
  knn<<<1, 1>>>(d_data, d_labels, n, dim, d_output, k, d_target);

  // wait for kernel to finish
  hipDeviceSynchronize();

  // check for errors
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    cout << "Error: " << hipGetErrorString(error) << endl;
    return 1;
  }

  // allocate memory for output
  double* output = (double*)malloc(sizeof(double) * k * dim);

  // copy output back to host
  hipMemcpy(output, d_output, sizeof(double*) * k, hipMemcpyDeviceToHost);

  // free device memory
  hipFree(d_data);
  hipFree(d_labels);
  hipFree(d_output);

  // Write output
  write_data(output_file, output);

  // Free memory
  free(data);

  return 0;
}