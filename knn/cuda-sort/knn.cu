#include "hip/hip_runtime.h"
#include "./knn.h"

int k = 0, n = 0, dim = 0;

void bubbleSortResult(double* output, double* target)
{
    for (int i = 0; i < k; i++) {
        for (int j = i + 1; j < k; j++) {
            double dist1 = 0;
            double dist2 = 0;
            for (int l = 0; l < dim; l++) {
                dist1 += (output[i * dim + l] - target[l]) * (output[i * dim + l] - target[l]);
                dist2 += (output[j * dim + l] - target[l]) * (output[j * dim + l] - target[l]);
            }
            if (dist1 > dist2) {
                for (int l = 0; l < dim; l++) {
                    double temp = output[i * dim + l];
                    output[i * dim + l] = output[j * dim + l];
                    output[j * dim + l] = temp;
                }
            }
        }
    }

    // print sorted output
    cout << "Sorted output:" << endl;
    for (int i = 0; i < k; i++) {
        cout << "Data " << i << ": ";
        for (int j = 0; j < dim; j++) {
            cout << fixed;
            cout.precision(10);
            cout << output[i * dim + j] << " ";
        }

        cout << "Distance: ";
        double dist = 0;
        for (int j = 0; j < dim; j++) {
            dist += (output[i * dim + j] - target[j]) * (output[i * dim + j] - target[j]);
        }
        cout << fixed;
        cout.precision(10);
        cout << sqrt(dist);
        cout << endl;
    }
}

int main(int argc, char** argv)
{
    if (argc != 3) {
        cout << "Usage: ./knn input_file output_file" << endl;
        return 1;
    }
    string input_file = argv[1];
    string output_file = argv[2];

    double* data = NULL;
    int* labels = NULL;
    double* target = NULL;

    // Read data
    read_data(input_file, data, labels, target);

    // Print top 5 data
    // print_top(data, labels, n, target);

    // allocate device memory
    double *d_data, *d_target;
    int* d_labels;

    hipMalloc(&d_data, sizeof(double) * n * dim);
    hipMalloc(&d_labels, sizeof(int) * n);
    hipMalloc(&d_target, sizeof(double) * dim);

    // copy data to device
    hipMemcpy(d_data, data, sizeof(double) * n * dim, hipMemcpyHostToDevice);
    hipMemcpy(d_labels, labels, sizeof(int) * n, hipMemcpyHostToDevice);
    hipMemcpy(d_target, target, sizeof(double) * dim, hipMemcpyHostToDevice);

    // call merge sort kernel
    bubbleSort<<<100, 1>>>(d_data, d_labels, n, dim, d_target, 2);

    // sync
    hipDeviceSynchronize();

    // copy data back to host
    hipMemcpy(data, d_data, sizeof(double) * n * dim, hipMemcpyDeviceToHost);

    printf("==\n");
    //  print data and distance
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < dim; j++) {
            printf("%f ", data[i * dim + j]);
        }
        // print distance
        double dist = 0;
        for (int j = 0; j < dim; j++) {
            dist += (data[i * dim + j] - target[j]) * (data[i * dim + j] - target[j]);
        }
        printf("Distance: %f\n", sqrt(dist));
    }
    printf("++++++++++++++++++++++++++++++\n");
    int blockSize = 1;
    int numBlocks = 3;
    int elementsPerThread = 4;
    long long sortedSize = 2;
    while (sortedSize < n) {
        mergeSort<<<numBlocks, blockSize, 2 * n * dim * sizeof(double) + 2 * sizeof(long long)>>>(d_data, d_labels, d_target, n, dim, elementsPerThread, sortedSize);
        hipDeviceSynchronize();
        sortedSize *= 2;
        printf("== sortedSize: %lld\n", sortedSize);
        printArr<<<1, 1>>>(d_data, n, dim, d_target);
    }

    // check for errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        cout << "Error: " << hipGetErrorString(error) << endl;
        return 1;
    }

    // allocate memory for output
    double* output = (double*)malloc(sizeof(double) * k * dim);
    int* labelsOutput = (int*)malloc(sizeof(int) * k);

    // copy output from device to host

    // sort output
    // bubbleSortResult(output, target);

    // free device memory
    hipFree(d_data);
    hipFree(d_labels);

    // Write output
    write_data(output_file, output, labelsOutput);

    // Free memory
    free(data);
    free(labels);
    free(target);
    free(output);
    free(labelsOutput);

    return 0;
}