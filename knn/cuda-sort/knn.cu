#include "hip/hip_runtime.h"
#include "./knn.h"

int k = 0, n = 0, dim = 0;

void bubbleSortResult(float* output, float* target)
{
    for (int i = 0; i < k; i++) {
        for (int j = i + 1; j < k; j++) {
            float dist1 = 0;
            float dist2 = 0;
            for (int l = 0; l < dim; l++) {
                dist1 += (output[i * dim + l] - target[l]) * (output[i * dim + l] - target[l]);
                dist2 += (output[j * dim + l] - target[l]) * (output[j * dim + l] - target[l]);
            }
            if (dist1 > dist2) {
                for (int l = 0; l < dim; l++) {
                    float temp = output[i * dim + l];
                    output[i * dim + l] = output[j * dim + l];
                    output[j * dim + l] = temp;
                }
            }
        }
    }

    // print sorted output
    cout << "Sorted output:" << endl;
    for (int i = 0; i < k; i++) {
        cout << "Data " << i << ": ";
        for (int j = 0; j < dim; j++) {
            cout << fixed;
            cout.precision(10);
            cout << output[i * dim + j] << " ";
        }

        cout << "Distance: ";
        float dist = 0;
        for (int j = 0; j < dim; j++) {
            dist += (output[i * dim + j] - target[j]) * (output[i * dim + j] - target[j]);
        }
        cout << fixed;
        cout.precision(10);
        cout << sqrt(dist);
        cout << endl;
    }
}

int main(int argc, char** argv)
{
    if (argc != 3) {
        cout << "Usage: ./knn input_file output_file" << endl;
        return 1;
    }
    string input_file = argv[1];
    string output_file = argv[2];

    float* data = NULL;
    int* labels = NULL;
    float* target = NULL;

    // Read data
    read_data(input_file, data, labels, target);

    // Print top 5 data
    // print_top(data, labels, n, target);

    // allocate device memory
    float *d_data, *d_target;
    int* d_labels;

    hipMalloc(&d_data, sizeof(float) * n * dim);
    hipMalloc(&d_labels, sizeof(int) * n);
    hipMalloc(&d_target, sizeof(float) * dim);

    // copy data to device
    hipMemcpy(d_data, data, sizeof(float) * n * dim, hipMemcpyHostToDevice);
    hipMemcpy(d_labels, labels, sizeof(int) * n, hipMemcpyHostToDevice);
    hipMemcpy(d_target, target, sizeof(float) * dim, hipMemcpyHostToDevice);
    long long sortedSize = 30;
    long long bNumThreads = 32;
    long long bNumBlocks = (n + sortedSize - 1) / (sortedSize);
    // call merge sort kernel
    bubbleSort<<<bNumBlocks, bNumThreads>>>(d_data, d_labels, n, dim, d_target, sortedSize);

    // sync
    hipDeviceSynchronize();

    // copy data back to host
    hipMemcpy(data, d_data, sizeof(float) * n * dim, hipMemcpyDeviceToHost);

    //  print data and distance
    // for (int i = 0; i < 5; i++) {
    //     for (int j = 0; j < dim; j++) {
    //         printf("%f ", data[i * dim + j]);
    //     }
    //     // print distance
    //     float dist = 0;
    //     for (int j = 0; j < dim; j++) {
    //         dist += (data[i * dim + j] - target[j]) * (data[i * dim + j] - target[j]);
    //     }
    //     printf("Distance: %f\n", sqrt(dist));
    // }
    // printf("++++++++++++++++++++++++++++++\n");
    long long elementsPerThread = 50;
    long long numThreads = 32;
    long long numBlocks = (n + sortedSize * 2 - 1) / (sortedSize * 2);
    long long elementsPerBlock = numThreads * elementsPerThread;
    while (sortedSize < n) {
        mergeSort<<<numBlocks, numThreads, 2 * elementsPerBlock * dim * sizeof(float) + 2 * sizeof(long long)>>>(d_data, d_labels, d_target, n, dim, elementsPerThread, sortedSize);
        hipError_t cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            printf("Error: %s\n", hipGetErrorString(cudaStatus));
        }
        sortedSize *= 2;
        printf("== sortedSize: %lld\n", sortedSize);
        // printArr<<<1, 1>>>(d_data, 5, dim, d_target);
        // break;
    }

    // check for errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        cout << "Error: " << hipGetErrorString(error) << endl;
        return 1;
    }

    // allocate memory for output
    float* output = (float*)malloc(sizeof(float) * k * dim);
    int* labelsOutput = (int*)malloc(sizeof(int) * k);

    // copy output from device to host
    hipMemcpy(output, d_data, sizeof(float) * k * dim, hipMemcpyDeviceToHost);

    // print top k data
    for (int i = 0; i < k; i++) {
        cout << "Data " << i << ": ";
        for (int j = 0; j < dim; j++) {
            cout << fixed;
            cout.precision(10);
            cout << output[i * dim + j] << " ";
        }

        cout << "Distance: ";
        float dist = 0;
        for (int j = 0; j < dim; j++) {
            dist += (output[i * dim + j] - target[j]) * (output[i * dim + j] - target[j]);
        }
        cout << fixed;
        cout.precision(10);
        cout << sqrt(dist);
        cout << endl;
    }

    // free device memory
    hipFree(d_data);
    hipFree(d_labels);

    // Write output
    write_data(output_file, output, labelsOutput);

    // Free memory
    free(data);
    free(labels);
    free(target);
    free(output);
    free(labelsOutput);

    return 0;
}