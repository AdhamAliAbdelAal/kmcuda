#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#include <string>
#include <iostream>
#include <cstdio>
#include <vector>
using namespace std;

#define MAX_ERR 1e-6
#define MAX_CLUSTERS 1000
#define LABELING_BLOCK_SIZE 1024
#define UPDATE_BLOCK_SIZE 256

__device__ float distance(float *point, float *centroid, int nDimensions){
    float sum = 0;
    for(int i = 0; i < nDimensions; i++){
        sum += (point[i] - centroid[i]) * (point[i] - centroid[i]);
    }
    return sqrt(sum);
}

__global__ void ICDKernel(float *centroids, float* ICD, int nDimensions, int nCentroids){
    // assumes the centroids can be fit in the shared memory and this kernel uses only one block
    extern __shared__ float centroids_shared[];
    // usage of blockIdx is redundant as this kernel is designed to be used by only one block
    // index = threadIdx.x
    int x = threadIdx.x;
    int y = threadIdx.y;
    // printf("x= %d, y= %d\n", x,y);
    // load the centroids to shared memory
    for(int j = x; j < nDimensions; j+=blockDim.x){
            centroids_shared[y*nDimensions+j] = centroids[y*nDimensions+j];
    }
    __syncthreads();
    // calculate the distance between the centroids
    float temp = distance(centroids_shared + x * nDimensions, centroids_shared + y * nDimensions, nDimensions);
    // write the distance to the global memory
    ICD[y*nCentroids+x] = temp;
}

// __global__ void RIDKernel(float *ICD, float *RID, int nCentroids){
//     // assumes the ICD can be fit in the shared memory and this kernel uses only one block
//     extern __shared__ float ICD_shared[];
//     // for each centroid we need to sort other centroids from closest to farthest and write that in RID

// }

__global__ void labelingKernel(float *points, float *centroids, float* currentCentroids, int *labels, int *counts, float *ICD, int nPoints, int nDimensions, int nCentroids){
    extern __shared__ float centroids_shared[];
    float * ICD_shared = centroids_shared + nCentroids*nDimensions;
    __shared__ int counts_privatization[MAX_CLUSTERS];
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    // printf("threadIdx: %d\n", threadIdx.x);
    // printf("blockdim: %d\n", blockDim.x);
    // printf("index: %d\n", index);
    // initialize centroids shared memory
    // int loadsPerThread = ceil((float)nCentroids*nDimensions/blockDim.x);
    for(int i=tid;i<nCentroids*nDimensions;i+=blockDim.x){
        centroids_shared[i] = currentCentroids[i];
        // printf("centroids_shared[%d]: %f\n", i, centroids_shared[i]);
    }

    // load ICD in ICD shared memory
    for(int i = tid; i < nCentroids*nCentroids; i+=blockDim.x){
        ICD_shared[i] = ICD[i];
    }

    // synchronize threads to ensure copying centroids
    __syncthreads();

    // copy point in register variable to avoid multiple access to global memory
    // float * point = new float[nDimensions]; 
    // float* point = (float*)malloc(nDimensions * sizeof(float));
    // for(int i = 0; i < nDimensions; i++){
    //     point[i] = points[index * nDimensions + i];
    // }  

    float * point = points + index * nDimensions;
    // start labeling
    int label = 0;
    if(index < nPoints){
        // int oldLabel = labels[index];
        // float oldDistance = distance(point, centroids_shared+oldLabel*nDimensions, nDimensions);
        // float minDistance =  oldDistance;
        // label = oldLabel;
        // // use register variable instead of global memory location
        // for(int i = 0; i < nCentroids; i++){
        //     if(i==oldLabel || ICD_shared[oldLabel*nCentroids+i] > 2*oldDistance){
        //         continue;
        //     }
        //     float d = distance(point, centroids_shared + i * nDimensions, nDimensions);
        //     if(d < minDistance){
        //         minDistance = d;
        //         label = i;
        //     }
        // }
        // write the value back to global memory
        label = index % nCentroids;
        labels[index] = label;
        // printf("Point %d: Label %d Distance %f\n", index, label, minDistance);
    }
    // to ensure that all threads calculate the distance before use centroids_shared as the shared memory for privatization
    __syncthreads();

    // initialize privatization arrays in each block
    for (int i = tid; i < nCentroids; i+=blockDim.x){
        counts_privatization[i] = 0;
    }
    // write reset the centroids in the shared memory
    for(int i = tid; i < nCentroids * nDimensions; i+=blockDim.x){
        centroids_shared[i] = 0;
    }
    // sync threads in the block is required to make sure the privatization arrays are initialized
    __syncthreads();

    // add the point to the centroid in the privatization array
    if(index < nPoints){
        // add 1 to the count of the label in the privatization array
        atomicAdd(counts_privatization + label, 1);
        // printf("point %d add to label %d\n", index, label);
        for(int i = 0; i < nDimensions; i++){
            atomicAdd(centroids_shared + label * nDimensions + i, point[i]);
        }
    }

    // sync threads to ensure that all threads have added the point to the privatization arrays
    __syncthreads();
    // write the privatization arrays back to global memory
    for(int i = tid; i < nCentroids; i+=blockDim.x){
        atomicAdd(counts + i, counts_privatization[i]);
    }
    for(int i = tid; i < nCentroids * nDimensions; i+=blockDim.x){
        // printf("centroids_shared[%d]: %f\n", i, centroids_shared[i]);
        atomicAdd(centroids + i, centroids_shared[i]);
    }
    // free the point
    // delete [] point;
}

__global__ void updateKernel(float *centroids, int *counts, float* oldCentroids, float *error, int nDimensions, int nCentroids){
    __shared__ float error_shared[UPDATE_BLOCK_SIZE];
    // printf("hello");
    // [TODO] make the kernel more efficient by using shared memory for counts as it can be used many times in single block
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int tid  = threadIdx.x;
    // initialize the error shared memory
    error_shared[tid] = 0;
    __syncthreads();
    if(index < nCentroids * nDimensions){
        // each thread update one float in the centroids array
        // mean calculations
        float temp = centroids[index];
        // printf("temp before [%d]: %f\n",index, temp);
        int clusterCount = index / nDimensions;
        int cnt = counts[clusterCount];
        // temp = temp?temp:0;
        // cnt = cnt?cnt:1;
        // printf("count[%d]: %d, index %d\n", clusterCount,counts[clusterCount],index);
        temp /= cnt;
        // printf("centroids[%d]: %f\n", index, temp);
        centroids[index] = temp;
        error_shared[tid] = abs(temp - oldCentroids[index]);
        // printf("error_shared[%d]: %f\n", tid, error_shared[tid]);
        oldCentroids[index] = temp;
    }

    // reduction step on the error array
    for(int stride = blockDim.x / 2; stride > 0; stride >>= 1){
        __syncthreads();
        if(tid < stride){
            error_shared[tid] += error_shared[tid + stride];
        }
    }
    __syncthreads();
    if(tid == 0){
        // add the error of the current block global error
        // printf("error: %f\n", *error);
        // printf("error_shared[0]: %f\n", error_shared[0]);

        atomicAdd(error, error_shared[0]);
        // printf("*********************: %f\n", error_shared[0]);
    }
    
}


float * allocateMatrix(int n, int m) {
    float * matrix = (float *)malloc(n * m * sizeof(float));
    return matrix;
}


void freeMatrix(float * matrix) {
    free(matrix);
}

void readMatrix(FILE* file, float* A, int m, int n){
    for(int i = 0; i < m; i++){
        for(int j = 0; j < n; j++){
            fscanf(file, "%f", &A[i*n+j]);
        }
    }
}

void cudaErrorCheck(hipError_t error, string message) {
    // Check for kernel launch errors
    if (error != hipSuccess) {
        fprintf(stderr, "Error: %s in %s\n", hipGetErrorString(error), message.c_str());
        // print status code
        fprintf(stderr, "Status code: %d\n", error);
        exit(-1);
    }
}
void kmeans(float * points, float * &centroids, int * &labels,  int nPoints, int nDimensions, int nCentroids, int maxIters){
    // Device Data
    float *d_points, *d_centroids, *d_oldCentroids, *error_val, *ICD;
    int *d_labels, *d_counts;

    float time;
    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    // Allocate memory on GPU
    hipMalloc(&d_points, nPoints * nDimensions * sizeof(float));
    // cudaErrorCheck("cudaMalloc d_points"
    hipMalloc(&d_centroids, nCentroids * nDimensions * sizeof(float));
    // cudaErrorCheck("cudaMalloc d_centroids");
    hipMalloc(&d_oldCentroids, nCentroids * nDimensions * sizeof(float));
    // cudaErrorCheck("cudaMalloc d_oldCentroids");
    hipMalloc(&error_val, sizeof(float));
    // cudaErrorCheck("cudaMalloc error_val");
    hipMalloc(&d_labels, nPoints * sizeof(int));
    // cudaErrorCheck("cudaMalloc d_labels");
    hipMalloc(&d_counts, nCentroids * sizeof(int));
    // cudaErrorCheck("cudaMalloc d_counts");
    hipMalloc(&ICD, nCentroids*nCentroids*sizeof(float));

    // Copy data to GPU
    hipMemcpy(d_points, points, nPoints * nDimensions * sizeof(float), hipMemcpyHostToDevice);
    // cudaErrorCheck("cudaMemcpy d_points");
    hipMemcpy(d_centroids, centroids, nCentroids * nDimensions * sizeof(float), hipMemcpyHostToDevice);
    // cudaErrorCheck("cudaMemcpy d_centroids");
    hipMemcpy(d_oldCentroids, centroids, nCentroids * nDimensions * sizeof(float), hipMemcpyHostToDevice);
    // cudaErrorCheck("cudaMemcpy d_oldCentroids");

    // Launch Kernel
    int labelingThreadsPerBlock = LABELING_BLOCK_SIZE;
    int labelingBlocksPerGrid = (nPoints + labelingThreadsPerBlock - 1) / labelingThreadsPerBlock;

    int updateThreadsPerBlock = UPDATE_BLOCK_SIZE;
    int updateBlocksPerGrid = (nCentroids * nDimensions + updateThreadsPerBlock - 1) / updateThreadsPerBlock;
    dim3 ICDThreadsPerBlock(nCentroids,nCentroids,1);
    for(int i = 0; i < maxIters; i++){
        // initialize counts to 0
        hipMemset(d_counts, 0, nCentroids * sizeof(int));
        hipMemset(error_val, 0, sizeof(float));
        hipMemset(d_centroids, 0, nCentroids * nDimensions * sizeof(float));
        // cudaErrorCheck("cudaMemset d_counts");
        ICDKernel<<<1,ICDThreadsPerBlock,nCentroids*nDimensions*sizeof(float)>>>(d_oldCentroids, ICD, nDimensions, nCentroids);
        cudaErrorCheck(hipDeviceSynchronize(),"ICDKernel");
        // printf("Iteration %d\n", i);
        labelingKernel<<<labelingBlocksPerGrid, labelingThreadsPerBlock, (nCentroids* nDimensions + nCentroids * nCentroids) * sizeof(float)>>>(d_points, d_centroids, d_oldCentroids, d_labels, d_counts, ICD, nPoints, nDimensions, nCentroids);

        cudaErrorCheck(hipDeviceSynchronize(),"labelingKernel");

        // Update Centroids
        updateKernel<<<updateBlocksPerGrid, updateThreadsPerBlock>>>(d_centroids, d_counts, d_oldCentroids, error_val, nDimensions, nCentroids);
        // cudaDeviceSynchronize();
        cudaErrorCheck(hipDeviceSynchronize(),"updateKernel");
        // printf("updated\n");
        float error;
        hipMemcpy(&error, error_val, sizeof(float), hipMemcpyDeviceToHost);
        cudaErrorCheck(hipDeviceSynchronize(),"hipMemcpy error");
        // printf("Error: %f\n", error);
        // if(error < MAX_ERR){
        //     printf("Converged\n");
        //     break;
        // }
    }
    hipMemcpy(centroids, d_centroids, nCentroids * nDimensions * sizeof(float), hipMemcpyDeviceToHost);
    // cudaErrorCheck("cudaMemcpy centroids");
    hipMemcpy(labels, d_labels, nPoints * sizeof(int), hipMemcpyDeviceToHost);
    // cudaErrorCheck("cudaMemcpy labels");
    printf("Done\n");

    // Free memory
    hipFree(d_points);
    hipFree(d_centroids);
    hipFree(d_oldCentroids);
    hipFree(d_labels);
    hipFree(d_counts);
    hipFree(error_val);
    
    hipEventRecord(stop, 0) ;
    hipEventSynchronize(stop) ;
    hipEventElapsedTime(&time, start, stop);
    printf("Time to generate:  %3.1f s \n", time/1000);
}

FILE* openFile(char* filename, string mode){
    FILE* file = fopen(filename, mode.c_str());
    if (file == NULL){
        printf("Error: file not found\n");
        exit(1);
    }
    return file;
}

void readData(FILE* file, int& nPoints, int& nDimensions, int& nCentroids, int& maxIters, float*& points, float*& centroids){
    fscanf(file, "%d %d %d %d", &nPoints, &nDimensions, &nCentroids, &maxIters);
    centroids = allocateMatrix(nCentroids, nDimensions);
    points = allocateMatrix(nPoints, nDimensions);
    readMatrix(file, points, nPoints, nDimensions);
    int step = nPoints / nCentroids;
    for(int i = 0; i < nCentroids; i++){
        for(int j = 0; j < nDimensions; j++){
            centroids[i*nDimensions+j] = points[i*step+j];
        }
    }
}

void printData(int nPoints, int nDimensions, int nCentroids, int maxIters, float* points, float* centroids){
    printf("nPoints = %d, nDimensions = %d, nCentroids = %d, maxIters = %d\n", nPoints, nDimensions, nCentroids, maxIters);
    printf("Centroids:\n");
    for(int i = 0; i < nCentroids; i++){
        for(int j = 0; j < nDimensions; j++){
            printf("%f ", centroids[i*nDimensions+j]);
        }
        printf("\n");
    }
    printf("Points:\n");
    for(int i = 0; i < nPoints; i++){
        for(int j = 0; j < nDimensions; j++){
            printf("%f ", points[i*nDimensions+j]);
        }
        printf("\n");
    }
}

void writeData(FILE *file,float *centroids, int *labels, int nPoints, int nDimensions, int nCentroids){
    for(int i = 0; i < nCentroids; i++){
        for(int j = 0; j < nDimensions; j++){
            fprintf(file, "%f ", centroids[i*nDimensions+j]);
        }
        fprintf(file, "\n");
    }
    for(int i = 0; i < nPoints; i++){
        fprintf(file, "%d\n", labels[i]);
    }

}
int main(int argc, char *argv[]){
    if (argc != 3)
    {
        printf("Usage: %s <input file path> <output file path> \n", argv[0]);
        return 1;
    }
    char* inputFileName = argv[1];
    char* outputFilename = argv[2];
    FILE* inputFile = openFile(inputFileName, "r");
    FILE* outputFile = openFile(outputFilename, "w");
    int nPoints, nDimensions, nCentroids, maxIters;
    float* points, *centroids;
    readData(inputFile, nPoints, nDimensions, nCentroids, maxIters, points, centroids);
    // printData(nPoints, nDimensions, nCentroids, maxIters, points, centroids);
    int *labels = (int*)malloc(nPoints * sizeof(int));
    kmeans(points, centroids, labels, nPoints, nDimensions, nCentroids, maxIters);
    // write output
    writeData(outputFile, centroids, labels, nPoints, nDimensions, nCentroids);
    // free memory
    freeMatrix(points);
    freeMatrix(centroids);
    free(labels);
    fclose(inputFile);
    fclose(outputFile);
}